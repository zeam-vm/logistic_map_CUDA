
#include <hip/hip_runtime.h>
#include <stdio.h>
#include<stdlib.h>
#include <math.h>
#include <sys/time.h>

#define LOOP 10
#define P 6700417
#define MU 22
#define NUM_VALUES 0x2000000

#define INLINE

static int logisticsmap_calc(int x, int p, int mu) {
    return mu * x * (x + 1) % p;
}

static int logisticsmap_loopCalc(int num, int x, int p, int mu) {
    for(int i = 0; i < num; i++) {
        x = logisticsmap_calc(x, p, mu);
    }
    return x;
}

#ifdef INLINE
__global__
void logisticsmap(int *x, int *p, int *mu, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
}
#else
__global__
void logisticsmap(int *x, int *p, int *mu, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j = 0; j < LOOP; j++) {
    	x[i] = mu[i] * x[i] * (x[i] + 1) % p[i];
    }
}
#endif

static void EC(hipError_t err, const char *message) {
	if(err != hipSuccess) {
		fprintf(stderr, "error in %s\n", message);
		exit(EXIT_FAILURE);
	}
}

int main(void){
    int *x, *p, *mu, *d_x, *d_p, *d_mu;
    const size_t n_byte = NUM_VALUES * sizeof(float);

    x = (int *)malloc(n_byte);
    p = (int *)malloc(n_byte);
    mu = (int *)malloc(n_byte);

    for(int i = 0; i < NUM_VALUES; i++) {
    	x[i] = i;
    	p[i] = P;
    	mu[i] = MU;
    }

    printf("start cudaMalloc\n");
    EC(hipMalloc((void**)&d_x, NUM_VALUES), "hipMalloc");
    EC(hipMalloc((void**)&d_p, NUM_VALUES), "hipMalloc");
    EC(hipMalloc((void**)&d_mu, NUM_VALUES), "hipMalloc");
    printf("finish cudaMalloc\n");

    printf("%p\n", d_x);

    printf("start cudaMemcpy\n");
    EC(hipMemcpy(d_x, x, n_byte, hipMemcpyHostToDevice), "cudaMemcpy: HostToDevice");
    EC(hipMemcpy(d_p, p, n_byte, hipMemcpyHostToDevice), "cudaMemcpy: HostToDevice");
    EC(hipMemcpy(d_mu, mu, n_byte, hipMemcpyHostToDevice), "cudaMemcpy: HostToDevice");
    printf("finish cudaMemcpy\n");

    printf("start kernel function\n");
    logisticsmap<<<(NUM_VALUES+255)/256, 256>>>(d_x, d_p, d_mu, n_byte);
    printf("finish kernel function\n");
    EC(hipMemcpy(x, d_x, n_byte, hipMemcpyDeviceToHost), "cudaMemcpy: DeviceToHost");

    EC(hipFree(d_x), "hipFree");
    EC(hipFree(d_p), "hipFree");
    EC(hipFree(d_mu), "hipFree");

    for(int i = 0; i < NUM_VALUES; i++) {
    	int expected = logisticsmap_loopCalc(10, i, p[i], mu[i]);
    	if(expected != x[i]) {
    		printf("invalid value of %d, expeted %d but %d\n", i, expected, x[i]);
    		return EXIT_FAILURE;
    	}
    }
    return EXIT_SUCCESS;
}
